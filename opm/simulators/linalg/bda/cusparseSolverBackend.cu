#include "hip/hip_runtime.h"
/*
  Copyright 2019 Big Data Accelerate

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef __NVCC__
	#error "Cannot compile for cusparse: NVIDIA compiler not found"
#endif

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

#include <opm/simulators/linalg/bda/cusparseSolverBackend.hpp>
#include <opm/simulators/linalg/bda/BdaResult.hpp>
#include <opm/simulators/linalg/bda/cuda_header.h>

#include "hipblas.h"
#include "hipsparse.h"
// For more information about cusparse, check https://docs.nvidia.com/cuda/cusparse/index.html

namespace Opm
{

	const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	const hipsparseOperation_t operation  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const hipsparseDirection_t order = HIPSPARSE_DIRECTION_ROW;

	double second(void){
		struct timeval tv;
		gettimeofday(&tv, nullptr);
		return (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
	}

	cusparseSolverBackend::cusparseSolverBackend(int verbosity_, int maxit_, double tolerance_) : verbosity(verbosity_), maxit(maxit_), tolerance(tolerance_), minit(0){
	}

	cusparseSolverBackend::~cusparseSolverBackend(){
		finalize();
	}

	// return true iff converged
	bool cusparseSolverBackend::gpu_pbicgstab(BdaResult& res){
		double t_total1, t_total2;
		int n = N;
		double rho = 1.0, rhop;
		double alpha, nalpha, beta;
		double omega, nomega, tmp1, tmp2;
		double norm, norm_0;
		double zero = 0.0;
		double one  = 1.0;
		double mone = -1.0;
		float it;

		t_total1 = second();

		hipsparseDbsrmv(cusparseHandle, order, operation, Nb, Nb, nnzb, &one, descr_M, d_bVals, d_bRows, d_bCols, BLOCK_SIZE, d_x, &zero, d_r);

		hipblasDscal(cublasHandle, n, &mone, d_r, 1);
		hipblasDaxpy(cublasHandle, n, &one, d_b, 1, d_r, 1);
		hipblasDcopy(cublasHandle, n, d_r, 1, d_rw, 1);
		hipblasDcopy(cublasHandle, n, d_r, 1, d_p, 1); 
		hipblasDnrm2(cublasHandle, n, d_r, 1, &norm_0);

		if(verbosity > 1){
			printf("Initial norm: %.5e\n", norm_0);
		}

		for(it = 0.5; it < maxit; it+=0.5){
			rhop = rho;
			hipblasDdot(cublasHandle, n, d_rw, 1, d_r, 1, &rho);

			if(it > 1){
				beta = (rho/rhop) * (alpha/omega);
				nomega = -omega;
				hipblasDaxpy(cublasHandle, n, &nomega, d_v, 1, d_p, 1);
				hipblasDscal(cublasHandle, n, &beta, d_p, 1);
				hipblasDaxpy(cublasHandle, n, &one, d_r, 1, d_p, 1);
			}

			// apply ilu0
			hipsparseDbsrsv2_solve(cusparseHandle, order, \
				operation, Nb, nnzb, &one, \
				descr_L, d_mVals, d_mRows, d_mCols, BLOCK_SIZE, info_L, d_p, d_t, policy, d_buffer);
			hipsparseDbsrsv2_solve(cusparseHandle, order, \
				operation, Nb, nnzb, &one, \
				descr_U, d_mVals, d_mRows, d_mCols, BLOCK_SIZE, info_U, d_t, d_pw, policy, d_buffer);

			// spmv
			hipsparseDbsrmv(cusparseHandle, order, \
				operation, Nb, Nb, nnzb, \
				&one, descr_M, d_bVals, d_bRows, d_bCols, BLOCK_SIZE, d_pw, &zero, d_v);

			hipblasDdot(cublasHandle, n, d_rw, 1, d_v, 1, &tmp1);
			alpha = rho / tmp1;
			nalpha = -alpha;
			hipblasDaxpy(cublasHandle, n, &nalpha, d_v, 1, d_r, 1);
			hipblasDaxpy(cublasHandle, n, &alpha, d_pw, 1, d_x, 1);
			hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);

			if(norm < tolerance * norm_0 && it > minit){
				break;
			}

			it += 0.5;

			// apply ilu0
			hipsparseDbsrsv2_solve(cusparseHandle, order, \
				operation, Nb, nnzb, &one, \
				descr_L, d_mVals, d_mRows, d_mCols, BLOCK_SIZE, info_L, d_r, d_t, policy, d_buffer);
			hipsparseDbsrsv2_solve(cusparseHandle, order, \
				operation, Nb, nnzb, &one, \
				descr_U, d_mVals, d_mRows, d_mCols, BLOCK_SIZE, info_U, d_t, d_s, policy, d_buffer);

			// spmv
			hipsparseDbsrmv(cusparseHandle, order, \
				operation, Nb, Nb, nnzb, &one, descr_M, \
				d_bVals, d_bRows, d_bCols, BLOCK_SIZE, d_s, &zero, d_t);

			hipblasDdot(cublasHandle, n, d_t, 1, d_r, 1, &tmp1);
			hipblasDdot(cublasHandle, n, d_t, 1, d_t, 1, &tmp2);
			omega = tmp1 / tmp2;
			nomega = -omega;
			hipblasDaxpy(cublasHandle, n, &omega, d_s, 1, d_x, 1);
			hipblasDaxpy(cublasHandle, n, &nomega, d_t, 1, d_r, 1);

			hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);


			if(norm < tolerance * norm_0 && it > minit){
				break;
			}

			if(verbosity > 1){
				printf("it: %.1f, norm: %.5e\n", it, norm);
			}
		}

		t_total2 = second();

		res.iterations = std::min(it, (float)maxit);
		res.reduction = norm/norm_0;
		res.conv_rate  = static_cast<double>(pow(res.reduction,1.0/it));
		res.elapsed = t_total2 - t_total1;
		res.converged = (it != (maxit + 0.5));

		if(verbosity > 0){
			printf("=== converged: %d, conv_rate: %.2f, time: %f, time per iteration: %f, iterations: %.1f\n", res.converged, res.conv_rate, res.elapsed, res.elapsed/it, it);
		}
		return res.converged;
	}


	void cusparseSolverBackend::initialize(int N, int nnz, int dim){
		this->N = N;
		this->nnz = nnz;
		this->BLOCK_SIZE = dim;
		this->nnzb = nnz/BLOCK_SIZE/BLOCK_SIZE;
		Nb = (N + dim - 1) / dim;
		printf("Initializing GPU, N: %d, nnz: %d, Nb: %d\n", N, nnz, Nb);
		printf("Minit: %d, maxit: %d, tolerance: %.1e\n", minit, maxit, tolerance);

		int deviceID = 0;
		hipSetDevice(deviceID);
		cudaCheckLastError("Could not get device");
		struct hipDeviceProp_t props;
		hipGetDeviceProperties(&props, deviceID);
		cudaCheckLastError("Could not get device properties");
		std::cout << "Name: " << props.name << "\n";
		printf("CC: %d.%d\n", props.major, props.minor);

		hipStreamCreate(&stream);
		cudaCheckLastError("Could not create stream");

		hipblasCreate(&cublasHandle);
		cudaCheckLastError("Could not create cublasHandle");

		hipsparseCreate(&cusparseHandle);
		cudaCheckLastError("Could not create cusparseHandle");

		hipMalloc((void**)&d_x, sizeof(double) * N);
		hipMalloc((void**)&d_b, sizeof(double) * N);
		hipMalloc((void**)&d_r, sizeof(double) * N);
		hipMalloc((void**)&d_rw,sizeof(double) * N);
		hipMalloc((void**)&d_p, sizeof(double) * N);
		hipMalloc((void**)&d_pw,sizeof(double) * N);
		hipMalloc((void**)&d_s, sizeof(double) * N);
		hipMalloc((void**)&d_t, sizeof(double) * N);
		hipMalloc((void**)&d_v, sizeof(double) * N);
		hipMalloc((void**)&d_bVals, sizeof(double) * nnz);
		hipMalloc((void**)&d_bCols, sizeof(double) * nnz);
		hipMalloc((void**)&d_bRows, sizeof(double) * (Nb+1));
		hipMalloc((void**)&d_mVals, sizeof(double) * nnz);
		cudaCheckLastError("Could not allocate enough memory on GPU");

		hipblasSetStream(cublasHandle, stream);
		cudaCheckLastError("Could not set stream to cublas");
		hipsparseSetStream(cusparseHandle, stream);
		cudaCheckLastError("Could not set stream to cusparse");

		hipHostMalloc((void**)&x, sizeof(double) * N);
		cudaCheckLastError("Could not allocate pinned host memory");

		initialized = true;
	} // end initialize()

	void cusparseSolverBackend::finalize(){
		hipFree(d_x);
		hipFree(d_b);
		hipFree(d_r);
		hipFree(d_rw);
		hipFree(d_p);
		hipFree(d_pw);
		hipFree(d_s);
		hipFree(d_t);
		hipFree(d_v);
		hipFree(d_mVals);
		hipFree(d_bVals);
		hipFree(d_bCols);
		hipFree(d_bRows);
		hipFree(d_buffer);
		hipsparseDestroyBsrilu02Info(info_M);
		hipsparseDestroyBsrsv2Info(info_L);
		hipsparseDestroyBsrsv2Info(info_U);
		hipsparseDestroyMatDescr(descr_B);
		hipsparseDestroyMatDescr(descr_M);
		hipsparseDestroyMatDescr(descr_L);
		hipsparseDestroyMatDescr(descr_U);
		hipsparseDestroy(cusparseHandle);
		hipblasDestroy(cublasHandle);
		hipHostUnregister(vals);
		hipHostUnregister(cols);
		hipHostUnregister(rows);
		hipStreamDestroy(stream);
		hipHostFree(x);
	} // end finalize()


	void cusparseSolverBackend::copy_system_to_gpu(double *vals, int *rows, int *cols, double *b){

		double t1, t2;
		if(verbosity > 2){
			t1 = second();
		}

		// information hipHostRegister: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1ge8d5c17670f16ac4fc8fcb4181cb490c
		// possible flags for hipHostRegister: hipHostRegisterDefault, hipHostRegisterPortable, hipHostRegisterMapped, hipHostRegisterIoMemory
		hipHostRegister(vals, nnz * sizeof(double), hipHostRegisterDefault);
		hipHostRegister(cols, nnz * sizeof(int), hipHostRegisterDefault);
		hipHostRegister(rows, (Nb+1) * sizeof(int), hipHostRegisterDefault);
		hipMemcpyAsync(d_bVals, vals, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_bCols, cols, nnz * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_bRows, rows, (Nb+1) * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_b, b, N * sizeof(double), hipMemcpyHostToDevice, stream);
		hipMemsetAsync(d_x, 0, sizeof(double) * N, stream);

		this->vals = vals;
		this->cols = cols;
		this->rows = rows;

		if(verbosity > 2){
			t2 = second();
			printf("cusparseSolver::copy_system_to_gpu(): %f s\n", t2-t1);
		}
	} // end copy_system_to_gpu()


	// don't copy rowpointers and colindices, they stay the same
	void cusparseSolverBackend::update_system_on_gpu(double *vals, double *b){

		double t1, t2;
		if(verbosity > 2){
			t1 = second();
		}

		hipMemcpyAsync(d_bVals, vals, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_b, b, N * sizeof(double), hipMemcpyHostToDevice, stream);
		hipMemsetAsync(d_x, 0, sizeof(double) * N, stream);

		if(verbosity > 2){
			t2 = second();
			printf("cusparseSolver::update_system_on_gpu(): %f s\n", t2-t1);
		}
	} // end update_system_on_gpu()


	void cusparseSolverBackend::reset_prec_on_gpu(){
		hipMemcpyAsync(d_mVals, d_bVals, nnz  * sizeof(double), hipMemcpyDeviceToDevice, stream);
	}


	void cusparseSolverBackend::analyse_matrix(){

		int d_bufferSize_M, d_bufferSize_L, d_bufferSize_U, d_bufferSize;
		double t1, t2;

		if(verbosity > 2){
			t1 = second();
		}

		hipsparseCreateMatDescr(&descr_B);
		hipsparseCreateMatDescr(&descr_M);
		hipsparseSetMatType(descr_B, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
		const hipsparseIndexBase_t base_type = HIPSPARSE_INDEX_BASE_ZERO;		// matrices from Flow are base0

		hipsparseSetMatIndexBase(descr_B, base_type);
		hipsparseSetMatIndexBase(descr_M, base_type);

		hipsparseCreateMatDescr(&descr_L);
		hipsparseSetMatIndexBase(descr_L, base_type);
		hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
		hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

		hipsparseCreateMatDescr(&descr_U);
		hipsparseSetMatIndexBase(descr_U, base_type);
		hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
		hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
		cudaCheckLastError("Could not initialize matrix descriptions");

		hipsparseCreateBsrilu02Info(&info_M);
		hipsparseCreateBsrsv2Info(&info_L);
		hipsparseCreateBsrsv2Info(&info_U);
		cudaCheckLastError("Could not create analysis info");

		hipMemcpyAsync(d_bRows, rows, sizeof(int)*(Nb+1), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_bCols, cols, sizeof(int)*nnz, hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_bVals, vals, sizeof(double)*nnz, hipMemcpyHostToDevice, stream);

		hipsparseDbsrilu02_bufferSize(cusparseHandle, order, Nb, nnzb,
			descr_M, d_bVals, d_bRows, d_bCols, BLOCK_SIZE, info_M, &d_bufferSize_M);
		hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzb,
			descr_L, d_bVals, d_bRows, d_bCols, BLOCK_SIZE, info_L, &d_bufferSize_L);
		hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzb,
			descr_U, d_bVals, d_bRows, d_bCols, BLOCK_SIZE, info_U, &d_bufferSize_U);
		cudaCheckLastError();
		d_bufferSize = std::max(d_bufferSize_M, std::max(d_bufferSize_L, d_bufferSize_U));
		
		hipMalloc((void**)&d_buffer, d_bufferSize);

		// analysis of ilu LU decomposition
		hipsparseDbsrilu02_analysis(cusparseHandle, order, \
			Nb, nnzb, descr_B, d_bVals, d_bRows, d_bCols, \
			BLOCK_SIZE, info_M, policy, d_buffer);

		int structural_zero;
		hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
		if(HIPSPARSE_STATUS_ZERO_PIVOT == status){
			fprintf(stderr, "ERROR block U(%d,%d) is not invertible\n", structural_zero, structural_zero);
			fprintf(stderr, "cusparse fails when a block has a 0.0 on its diagonal, these should be replaced in BdaBridge::checkZeroDiagonal()\n");
			exit(1);
		}

		// analysis of ilu apply
		hipsparseDbsrsv2_analysis(cusparseHandle, order, operation, \
			Nb, nnzb, descr_L, d_bVals, d_bRows, d_bCols, \
			BLOCK_SIZE, info_L, policy, d_buffer);

		hipsparseDbsrsv2_analysis(cusparseHandle, order, operation, \
			Nb, nnzb, descr_U, d_bVals, d_bRows, d_bCols, \
			BLOCK_SIZE, info_U, policy, d_buffer);
		cudaCheckLastError("Could not analyse level information");

		if(verbosity > 2){
			t2 = second();
			printf("cusparseSolver::analyse_matrix(): %f s\n", t2-t1);
		}

	} // end analyse_matrix()

	bool cusparseSolverBackend::create_preconditioner(){

		double t1, t2;
		if(verbosity > 2){
			t1 = second();
		}

		d_mCols = d_bCols;
		d_mRows = d_bRows;
		hipsparseDbsrilu02(cusparseHandle, order, \
			Nb, nnzb, descr_M, d_mVals, d_mRows, d_mCols, \
			BLOCK_SIZE, info_M, policy, d_buffer);

		int structural_zero;
		hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
		if(HIPSPARSE_STATUS_ZERO_PIVOT == status){
			fprintf(stderr, "WARNING block U(%d,%d) is not invertible\n", structural_zero, structural_zero);
			fprintf(stderr, "cusparse fails when a block has a 0.0 on its diagonal, these should be replaced in BdaBridge::checkZeroDiagonal()\n");
			return false;
		}

		if(verbosity > 2){
			hipStreamSynchronize(stream);
			t2 = second();
			printf("cusparseSolver::create_preconditioner(): %f s\n", t2-t1);
		}
		return true;
	} // end create_preconditioner()


	// return true iff converged
	bool cusparseSolverBackend::solve_system(BdaResult &res){
		// actually solve
		bool converged = gpu_pbicgstab(res);
		hipStreamSynchronize(stream);
		cudaCheckLastError("Something went wrong during the GPU solve");
		return converged;
	} // end solve_system()


	// copy result to host memory
	// caller must be sure that x is a valid array
	void cusparseSolverBackend::post_process(double *x){

		double t1, t2;
		if(verbosity > 2){
			t1 = second();
		}

		hipMemcpyAsync(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost, stream);
		hipStreamSynchronize(stream);

		if(verbosity > 2){
			t2 = second();
			printf("cusparseSolver::post_process(): %f s\n", t2-t1);
		}
	} // end post_process()


	bool cusparseSolverBackend::isInitialized(){
		return initialized;
	}

}


